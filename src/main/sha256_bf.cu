#include "hip/hip_runtime.h"
#include <stdio.h>
#include "sha256.h"
#include "utils.h"
#include "combinations.h"
#include <chrono>
#include <string>
#include <stdlib.h> 

using namespace std;

__global__  
void sha256(BYTE * data, BYTE * hash, int len) {
	int j = (blockIdx.x * blockDim.x + threadIdx.x) * 32;
	SHA256_CTX ctx;
	sha256_init(&ctx);
	sha256_update(&ctx, data + j, len);
	sha256_final(&ctx, hash + j);
}

void usage(char * exe) {
	printf("%s blocks threads charset passlen char_limit target \n", exe);
}

int main(int argc, char ** argv) {	
	if (argc != 7) {
		usage(argv[0]);
		return 1;
	}

	int blocks =  atoi(argv[1]);
	int threads_per_block = atoi(argv[2]);
	string charset = argv[3];

	int pass_len = atoi(argv[4]);
	int char_limit = atoi(argv[5]);
	string target_pass = argv[6];

	int salt_len = 8;
	
	int N = blocks * threads_per_block;

	BYTE * hash;
	size_t hash_size = N * 32 * sizeof(BYTE);
	
	Combinations combinations(pass_len, charset , char_limit);
	
	BYTE encoded_pass_with_salt[40];
	BYTE salt[8];
	BYTE encoded_pass[32];

	string * clear_pass = new string[N];

	hexstr2byte(target_pass, encoded_pass_with_salt);
	memcpy(salt, encoded_pass_with_salt, salt_len);
	memcpy(encoded_pass, encoded_pass_with_salt + salt_len, 32);

	hipMallocManaged(&hash,  hash_size);
	
	printf("Start \n");
	for (unsigned long j = 0; j < 20000000000; j++) {

		auto begin = chrono::high_resolution_clock::now();
		for (int i = 0; i < N; i ++) {
			clear_pass[i] = combinations.next();
			memcpy(hash + (i * 32), salt, salt_len);
			memcpy(hash + salt_len + (i * 32), clear_pass[i].c_str(), pass_len);
		}


		sha256<<<blocks,threads_per_block>>>(hash, hash, salt_len + pass_len); 
		for (int i = 0; i < 1023; i ++) {
			sha256<<<blocks,threads_per_block>>>(hash, hash, 32); 
		}

		hipDeviceSynchronize();
	
		for (int i = 0; i < N; i++) {
			if (memcmp(hash + (i * 32), encoded_pass, 32 * sizeof(BYTE)) == 0) {
				printf("password found: {%s} \n", clear_pass[i].c_str());

				hipFree(hash);
				return 0;
			}
		}
		
		auto end = chrono::high_resolution_clock::now();
		double duration = chrono::duration<double>(end - begin).count();

		double pass_per_second = (N / duration);
		printf("current(%d): duration: %f s - last: %s - %f pass/sec \n", j, duration, clear_pass[0].c_str(), pass_per_second);
		
	}

	hipFree(hash);
    
    return 0;
}
