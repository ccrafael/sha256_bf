#include "hip/hip_runtime.h"
#include "utils.h"
#include <sstream>
#include <iomanip>
#include <bitset>


using namespace std;

 BYTE char2byte(char input) {
   if(input >= '0' && input <= '9') {
     return input - '0';
   } else if(input >= 'A' && input <= 'F') {
     return input - 'A' + 10;
   } else if(input >= 'a' && input <= 'f') {
         return input - 'a' + 10;
   }
   throw std::invalid_argument("Invalid input string");
 }
 
 // buffer need to have size at least ceil(str.len/2)
 void hexstr2byte(string str, BYTE * buffer) { 
         for (int i = 0; i < str.size(); i = i + 2) { 
                 *(buffer++) = char2byte(str[i])*16 + char2byte(str[i+1]);
         }
 }
 
 string byte2hexstr(BYTE *data, int len) {
      stringstream ss;
      ss << hex;
      
      for (int i = 0; i < len; ++i) {
          ss << setw(2) << setfill('0') << (int)data[i];
      }
      
      return ss.str();
 }


 

